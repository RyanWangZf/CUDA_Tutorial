#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc,char **argv){
    
    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // memory size
    unsigned int isize = 1<<22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);

    printf("%s starting at ",argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2f MB\n",dev,deviceProp.name,
        isize,nbytes/(1024.0f*1024.0f));

    // allocate pinned host memory
    float *h_a;
    hipHostMalloc((float**)&h_a,nbytes, hipHostMallocDefault);

    // allocate the device memory
    // Attention the different definition format between the malloc & cuadMalloc
    float *d_a;
    hipMalloc((float **)&d_a,nbytes);

    // initialize the host memory
    for(unsigned int i = 0; i < isize; i++) h_a[i] = .5f;

    // transfer data from the host to the device
    hipMemcpy(d_a,h_a,nbytes,hipMemcpyHostToDevice);

    // transfer data from device to the host
    hipMemcpy(h_a,d_a,nbytes,hipMemcpyDeviceToHost);

    // free
    hipFree(d_a);
    hipHostFree(h_a);

    // reset
    hipDeviceReset();
    return EXIT_SUCCESS;
    }

    
